#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
// 97 to 122 use only lowercase letters
// 65 to 90 use only capital letters
// 48 to 57 use only numbers

#define START_CHAR 48
#define END_CHAR 122
#define MAXIMUM_PASSWORD 20

__device__ __host__ long long my_pow(long long x, int y)
{
    long long res = 1;
    if (y == 0)
        return res;
    else
        return x * my_pow(x, y - 1);
}

__device__ int my_strlen(char *s)
{
    int len = 0;
    while (s[len] != '\0')
    {
        len = len + 1;
    }
    return (len);
}

__global__ void bruteForce(char *pass)
{

    int pass_b26[MAXIMUM_PASSWORD];

    long long int j = blockIdx.x * blockDim.x + threadIdx.x;
    long long int pass_decimal = 0;
    int base = END_CHAR - START_CHAR + 2;
    int size = my_strlen(pass);
    for (int i = 0; i < size; i++)
        pass_b26[i] = (int)pass[i] - START_CHAR + 1;

    for (int i = size - 1; i > -1; i--)
        pass_decimal += (long long int)pass_b26[i] * my_pow(base, i);

    long long int max = my_pow(base, size);
    char s[MAXIMUM_PASSWORD];

    for (; j < max; j += blockDim.x * gridDim.x)
    {
        if (j == pass_decimal)
        {
            printf("Found password!\n");
            int index = 0;

            printf("Password in decimal base: %lli\n", j);
            while (j > 0)
            {
                s[index++] = START_CHAR + j % base - 1;
                j /= base;
            }
            s[index] = '\0';
            printf("Found password: %s\n", s);
            break;
        }
    }
}

int main(int argc, char **argv)
{
    char password[MAXIMUM_PASSWORD], *pass_d;

    strcpy(password, argv[1]);
    time_t t1, t2;
    double dif, x, speedup;
    hipMalloc(&pass_d, sizeof(char) * MAXIMUM_PASSWORD);
    hipMemcpy(pass_d, password, sizeof(char) * MAXIMUM_PASSWORD, hipMemcpyHostToDevice);

    int deviceId, numberOfSMs;
    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
    int number_of_blocks = numberOfSMs * 32;
    int threads_per_block = atoi(argv[2]);

    time(&t1);
    printf("Try to broke the password: %s\n", password);
    bruteForce<<<number_of_blocks, threads_per_block>>>(pass_d);
    hipDeviceSynchronize();
    time(&t2);

    dif = difftime(t2, t1);
    FILE *fptr;
    FILE *fptr1;
    char c[1000];

    if ((fptr1 = fopen("firstValue.dat", "r")) != NULL)
    {
        fscanf(fptr1, "%[^\n]", c);
        x = atof(c);

        speedup = x / dif;

        fclose(fptr1);
    }

    if ((fptr = fopen("speedupCUDA.dat", "a+")) != NULL)
    {
        fprintf(fptr, "%d\t%1.2f\n", threads_per_block, speedup);
        fclose(fptr);
    }
    else
    {
        fopen("speedupCUDA.dat", "w+");
        fprintf(fptr, "%d\t%1.2f\n", threads_per_block, speedup);
        fclose(fptr);
    }
    printf("\n%1.2f seconds\n", dif);
    return 0;
}