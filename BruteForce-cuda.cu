#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
// 97 to 122 use only lowercase letters
// 65 to 90 use only capital letters
// 48 to 57 use only numbers

#define START_CHAR 97
#define END_CHAR 122
#define MAXIMUM_PASSWORD 20

__device__ __host__ long long my_pow(long long x, int y)
{
    long long res = 1;
    if (y == 0)
        return res;
    else
        return x * my_pow(x, y - 1);
}

__device__ int my_strlen(char *s)
{
    int len = 0;
    while (s[len] != '\0')
    {
        len = len + 1;
    }
    return (len);
}

__global__ void bruteForce(char *pass)
{
    int pass_b26[MAXIMUM_PASSWORD];

    long long int j = blockIdx.x * blockDim.x + threadIdx.x;
    long long int pass_decimal = 0;
    int base = END_CHAR - START_CHAR + 2;

    int size = my_strlen(pass);
    for (int i = 0; i < size; i++)
        pass_b26[i] = (int)pass[i] - START_CHAR + 1;

    for (int i = size - 1; i > -1; i--)
        pass_decimal += (long long int)pass_b26[i] * my_pow(base, i);

    long long int max = my_pow(base, size);
    char s[MAXIMUM_PASSWORD];

    for (; j < max; j += blockDim.x * gridDim.x)
    {
        if (j == pass_decimal)
        {
            printf("Found password!\n");
            int index = 0;

            printf("Password in decimal base: %lli\n", j);
            while (j > 0)
            {
                s[index++] = START_CHAR + j % base - 1;
                j /= base;
            }
            s[index] = '\0';
            printf("Found password: %s\n", s);
            break;
        }
    }
}

int main(int argc, char **argv)
{
    char password[MAXIMUM_PASSWORD], *pass_d;
    struct timespec tstart = {0, 0}, tend = {0, 0};
    strcpy(password, argv[1]);
    time_t t1, t2;
    double dif;
    hipMalloc(&pass_d, sizeof(char) * MAXIMUM_PASSWORD);
    hipMemcpy(pass_d, password, sizeof(char) * MAXIMUM_PASSWORD, hipMemcpyHostToDevice);

    int deviceId, numberOfSMs;
    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
    int number_of_blocks = numberOfSMs * 32;
    int threads_per_block = 1024;

    time(&t1);
    clock_gettime(CLOCK_MONOTONIC, &tstart);
    printf("Try to broke the password: %s\n", password);
    bruteForce<<<number_of_blocks, threads_per_block>>>(pass_d);
    hipDeviceSynchronize();
    clock_gettime(CLOCK_MONOTONIC, &tend);
    time(&t2);

    dif = difftime(t2, t1);

    printf("\n %.5f seconds\n",
           ((double)tend.tv_sec + 1.0e-9 * tend.tv_nsec) -
               ((double)tstart.tv_sec + 1.0e-9 * tstart.tv_nsec));

    return 0;
}